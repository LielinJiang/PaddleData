#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "image_decoder.h"

template <typename T>
__global__ void copy(const T* in,
                           T* out,
                           const int numel) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < numel;
       index += blockDim.x * gridDim.x) {
    out[index] = in[index];
  }

}

template <typename T>
void copy_kernelLauncher(const float* in,
                               T* out,
                               const int numel,
                               hipStream_t stream){
  dim3 grid(64);
  dim3 block(64);
  copy<<<grid, block, 0, stream>>>(in, out, numel);
}
